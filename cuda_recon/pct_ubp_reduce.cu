
#include <hip/hip_runtime.h>
/**
 * @file pct_ubp.cu
 * 
 * CUDA code to perform ubp on GPU
 */



/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void ubp( 
                      float * out,
                      const float * x, 
                      const float * y, 
                      const float * z,
                      const float * xk, 
                      const float * yk, 
                      const float * zk,
					  const float * xe,
					  const float * ye,
                      const unsigned int Nx,
                      const unsigned int log2Nx,
                      const unsigned int Nphi, 
                      const unsigned int Nt,
                      const float fixedDelay, 
                      const float tv, 
                      const float * data ) {
    // Work out which thread we are

    __shared__ float sample[512];
            
    int xIdx = blockIdx.x &(Nx-1);
    int angleIdx = blockIdx.x >> log2Nx;
    int transducerIdx = angleIdx * blockDim.x + threadIdx.x; 
    int globalVolumeIndex = xIdx + blockIdx.y * Nx + blockIdx.z * Nx * gridDim.y;
    
    
    // Get our X and Y coords
    float const xi = x[xIdx];
    float const yi = y[blockIdx.y];
    float const zi = z[blockIdx.z];


    //unsigned long start = 0;

    float const xki = xk[transducerIdx];
    float const yki = yk[transducerIdx];
    float const zki = zk[transducerIdx];
	
	float const xei = xe[angleIdx];
    float const yei = ye[angleIdx];
    
	float const d = sqrt( (xi-xki) * (xi-xki) + (yi-yki) * (yi-yki) + (zi-zki) * (zi-zki));
	float const d_emit = sqrt( (xi-xei) * (xi-xei) + (yi-yei) * (yi-yei) );
    float idxf = rintf( (d + d_emit)* tv) - fixedDelay;
    int idx = __float2int_rd(idxf);
    idx = max( idx, 0);
    idx = min( idx, Nt - 1); 
    idx = threadIdx.x + blockDim.x*idx + blockDim.x*Nt*angleIdx;
    //idx = 0;
    sample[threadIdx.x] = data[idx];
    __syncthreads();
	int nTotalThreads = blockDim.x;	// Total number of active threads

	while(nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
		if (threadIdx.x < halfPoint)
		{
			
			// when calculating the average, sum and divide
			sample[threadIdx.x] += sample[threadIdx.x + halfPoint];
		}
		__syncthreads();

		nTotalThreads = (nTotalThreads >> 1);	// divide by two.
	}

	// At this point in time, thread zero has the min, max, and average
	// It's time for thread zero to write it's final results.
	// Note that the address structure of pResults is different, because
	// there is only one value for every thread block.

	if (threadIdx.x == 0)
	{
        atomicAdd(&out[globalVolumeIndex], sample[0]);
	}
    //atomicAdd(&out[globalBlockIndex],data[idx]);

            
}
